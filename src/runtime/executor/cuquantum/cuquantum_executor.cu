
#include <hip/hip_runtime.h>
/** ExaTN: Tensor Runtime: Tensor network executor: NVIDIA cuQuantum
REVISION: 2021/12/27

Copyright (C) 2018-2021 Dmitry Lyakh
Copyright (C) 2018-2021 Oak Ridge National Laboratory (UT-Battelle)

Rationale:

**/

#ifdef CUQUANTUM

#include <cutensornet.h>
#include <cutensor.h>
#include <cuda_runtime.h>

#include <vector>
#include <unordered_map>
#include <type_traits>

#include <iostream>

#include "talshxx.hpp"

#include "cuquantum_executor.hpp"


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != cudaSuccess ) \
{ printf("Error: %s in line %d\n", cudaGetErrorString(err), __LINE__); std::abort(); } \
};

#define HANDLE_CTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
{ printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); std::abort(); } \
};


namespace exatn {
namespace runtime {

struct TensorDescriptor {
 std::vector<int32_t> modes;
 std::vector<int64_t> extents;
};

struct TensorNetworkReq {
 std::shared_ptr<numerics::TensorNetwork> network;
 std::unordered_map<numerics::TensorHashType,TensorDescriptor> tensor_descriptors;
 std::unordered_map<int32_t,int64_t> index_extents;
 cutensornetNetworkDescriptor_t net_descriptor;
 cutensornetContractionOptimizerConfig_t opt_config;
 cutensornetContractionOptimizerInfo_t opt_info;
 cutensornetContractionPlan_t comp_plan;
 cudaStream_t stream;
};


CuQuantumExecutor::CuQuantumExecutor(TensorImplFunc tensor_data_access_func):
 tensor_data_access_func_(std::move(tensor_data_access_func))
{
 static_assert(std::is_same<cutensornetHandle_t,void*>::value,"#FATAL(exatn::runtime::CuQuantumExecutor): cutensornetHandle_t != (void*)");

 const size_t version = cutensornetGetVersion();
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): cuTensorNet backend version " << version << std::endl;

 int num_gpus = 0;
 auto error_code = talshDeviceCount(DEV_NVIDIA_GPU,&num_gpus); assert(error_code == TALSH_SUCCESS);
 for(int i = 0; i < num_gpus; ++i){
  if(talshDeviceState(i,DEV_NVIDIA_GPU) >= DEV_ON) gpus_.emplace_back(i);
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Number of available GPUs = " << gpus_.size() << std::endl;

 ctn_handles_.resize(gpus_.size());
 for(const auto & gpu_id: gpus_){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu_id));
  HANDLE_CTN_ERROR(cutensornetCreate((cutensornetHandle_t*)(&ctn_handles_[gpu_id])));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Created cuTensorNet contexts for all available GPUs" << std::endl;
}


CuQuantumExecutor::~CuQuantumExecutor()
{
 bool success = sync(); assert(success);
 for(const auto & gpu_id: gpus_){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu_id));
  HANDLE_CTN_ERROR(cutensornetDestroy((cutensornetHandle_t)(ctn_handles_[gpu_id])));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Destroyed cuTensorNet contexts for all available GPUs" << std::endl;
 ctn_handles_.clear();
 gpus_.clear();
}


int CuQuantumExecutor::execute(std::shared_ptr<numerics::TensorNetwork> network,
                               const TensorOpExecHandle exec_handle)
{
 int error_code = 0;
 //`Finish
 return error_code;
}


bool CuQuantumExecutor::executing(const TensorOpExecHandle exec_handle)
{
 auto iter = active_networks_.find(exec_handle);
 return (iter != active_networks_.end());
}


bool CuQuantumExecutor::sync(const TensorOpExecHandle exec_handle,
                             int * error_code,
                             bool wait)
{
 bool synced = true;
 *error_code = 0;
 auto iter = active_networks_.find(exec_handle);
 if(iter != active_networks_.end()){
  //`Finish
 }
 return synced;
}


bool CuQuantumExecutor::sync()
{
 bool synced = true;
 //`Finish
 return synced;
}

} //namespace runtime
} //namespace exatn

#endif //CUQUANTUM
