
#include <hip/hip_runtime.h>
/** ExaTN: Tensor Runtime: Tensor network executor: NVIDIA cuQuantum
REVISION: 2021/12/29

Copyright (C) 2018-2021 Dmitry Lyakh
Copyright (C) 2018-2021 Oak Ridge National Laboratory (UT-Battelle)

Rationale:

**/

#ifdef CUQUANTUM

#include <cutensornet.h>
#include <cutensor.h>
#include <cuda_runtime.h>

#include <vector>
#include <unordered_map>
#include <type_traits>

#include <iostream>

#include "talshxx.hpp"

#include "linear_memory.hpp"

#include "cuquantum_executor.hpp"


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != cudaSuccess ) \
{ printf("Error: %s in line %d\n", cudaGetErrorString(err), __LINE__); std::abort(); } \
};

#define HANDLE_CTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
{ printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); std::abort(); } \
};


namespace exatn {
namespace runtime {

struct TensorDescriptor {
 std::vector<int32_t> modes;   //indices associated with tensor dimensions
 std::vector<int64_t> extents; //tensor dimension extents
 std::vector<int64_t> strides; //tensor dimension strides (optional)
 void * body_ptr = nullptr;    //pointer to the tensor body image
 std::size_t volume = 0;       //tensor body volume
 cudaDataType_t data_type;     //tensor element data type
};

struct TensorNetworkReq {
 std::shared_ptr<numerics::TensorNetwork> network;
 std::unordered_map<numerics::TensorHashType,TensorDescriptor> tensor_descriptors;
 std::unordered_map<int32_t,int64_t> index_extents;
 cutensornetNetworkDescriptor_t net_descriptor;
 cutensornetContractionOptimizerConfig_t opt_config;
 cutensornetContractionOptimizerInfo_t opt_info;
 cutensornetContractionPlan_t comp_plan;
 cudaStream_t stream;
 cutensornetComputeType_t compute_type;
 void * memory_window_ptr = nullptr;
 TensorNetworkQueue::ExecStat exec_status = TensorNetworkQueue::ExecStat::Idle;
};


CuQuantumExecutor::CuQuantumExecutor(TensorImplFunc tensor_data_access_func):
 tensor_data_access_func_(std::move(tensor_data_access_func))
{
 static_assert(std::is_same<cutensornetHandle_t,void*>::value,"#FATAL(exatn::runtime::CuQuantumExecutor): cutensornetHandle_t != (void*)");

 const size_t version = cutensornetGetVersion();
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): cuTensorNet backend version " << version << std::endl;

 int num_gpus = 0;
 auto error_code = talshDeviceCount(DEV_NVIDIA_GPU,&num_gpus); assert(error_code == TALSH_SUCCESS);
 for(int i = 0; i < num_gpus; ++i){
  if(talshDeviceState(i,DEV_NVIDIA_GPU) >= DEV_ON){
   gpu_attr_.emplace_back(std::make_pair(i,DeviceAttr{}));
   gpu_attr_.back().second.workspace_ptr = talsh::getDeviceBufferBasePtr(DEV_NVIDIA_GPU,i);
   assert(reinterpret_cast<std::size_t>(gpu_attr_.back().second.workspace_ptr) % MEM_ALIGNMENT == 0);
   gpu_attr_.back().second.buffer_size = talsh::getDeviceMaxBufferSize(DEV_NVIDIA_GPU,i);
   std::size_t wrk_size = static_cast<float>(gpu_attr_.back().second.buffer_size) * WORKSPACE_FRACTION;
   wrk_size -= wrk_size % MEM_ALIGNMENT;
   gpu_attr_.back().second.workspace_size = wrk_size;
   gpu_attr_.back().second.buffer_size -= wrk_size;
   gpu_attr_.back().second.buffer_ptr = (void*)(((char*)(gpu_attr_.back().second.workspace_ptr)) + wrk_size);
  }
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Number of available GPUs = " << gpu_attr_.size() << std::endl;

 for(const auto & gpu: gpu_attr_){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu.first));
  HANDLE_CTN_ERROR(cutensornetCreate((cutensornetHandle_t*)(&gpu.second.cutn_handle)));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Created cuTensorNet contexts for all available GPUs" << std::endl;

 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): GPU configuration:\n";
 for(const auto & gpu: gpu_attr_){
  std::cout << " GPU #" << gpu.first
            << ": wrk_ptr = " << gpu.second.workspace_ptr
            << ", size = " << gpu.second.workspace_size
            << "; buf_ptr = " << gpu.second.buffer_ptr
            << ", size = " << gpu.second.buffer_size << std::endl;
 }
}


CuQuantumExecutor::~CuQuantumExecutor()
{
 bool success = sync(); assert(success);
 for(const auto & gpu: gpu_attr_){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu.first));
  HANDLE_CTN_ERROR(cutensornetDestroy((cutensornetHandle_t)(gpu.second.cutn_handle)));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Destroyed cuTensorNet contexts for all available GPUs" << std::endl;
 gpu_attr_.clear();
}


TensorNetworkQueue::ExecStat CuQuantumExecutor::execute(std::shared_ptr<numerics::TensorNetwork> network,
                                                        const TensorOpExecHandle exec_handle)
{
 assert(network);
 TensorNetworkQueue::ExecStat exec_stat = TensorNetworkQueue::ExecStat::None;
 auto res = active_networks_.emplace(std::make_pair(exec_handle, new TensorNetworkReq{}));
 if(res.second){
  auto tn_req = res.first->second;
  tn_req->network = network;
  exec_stat = tn_req->exec_status;
  //`Finish
 }else{
  std::cout << "#WARNING(exatn::runtime::CuQuantumExecutor): execute: Repeated tensor network submission detected!\n";
 }
 return exec_stat;
}


TensorNetworkQueue::ExecStat CuQuantumExecutor::sync(const TensorOpExecHandle exec_handle,
                                                     int * error_code,
                                                     bool wait)
{
 *error_code = 0;
 TensorNetworkQueue::ExecStat exec_stat = TensorNetworkQueue::ExecStat::None;
 auto iter = active_networks_.find(exec_handle);
 if(iter != active_networks_.end()){
  auto tn_req = iter->second;
  exec_stat = tn_req->exec_status;
  //`Finish
 }
 return exec_stat;
}


bool CuQuantumExecutor::sync()
{
 bool synced = true;
 //`Finish
 return synced;
}

} //namespace runtime
} //namespace exatn

#endif //CUQUANTUM
