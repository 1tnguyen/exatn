
#include <hip/hip_runtime.h>
/** ExaTN: Tensor Runtime: Tensor network executor: NVIDIA cuQuantum
REVISION: 2021/12/24

Copyright (C) 2018-2021 Dmitry Lyakh
Copyright (C) 2018-2021 Oak Ridge National Laboratory (UT-Battelle)

Rationale:

**/

#ifdef CUQUANTUM

#include <cutensornet.h>
#include <cutensor.h>
#include <cuda_runtime.h>

#include <vector>
#include <unordered_map>

#include <iostream>

#include "talshxx.hpp"

#include "cuquantum_executor.hpp"


#define HANDLE_CUDA_ERROR(x) \
{ const auto err = x; \
  if( err != cudaSuccess ) \
{ printf("Error: %s in line %d\n", cudaGetErrorString(err), __LINE__); std::abort(); } \
};

#define HANDLE_CTN_ERROR(x) \
{ const auto err = x; \
  if( err != CUTENSORNET_STATUS_SUCCESS ) \
{ printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); std::abort(); } \
};


namespace exatn {
namespace runtime {

struct TensorDescriptor {
 std::vector<int32_t> modes;
 std::vector<int64_t> extents;
};

struct TensorNetworkReq {
 std::shared_ptr<numerics::TensorNetwork> network;
 std::unordered_map<numerics::TensorHashType,TensorDescriptor> tensor_descriptors;
 std::unordered_map<int32_t,int64_t> index_extents;
 cutensornetNetworkDescriptor_t net_descriptor;
 cutensornetContractionOptimizerConfig_t opt_config;
 cutensornetContractionOptimizerInfo_t opt_info;
 cutensornetContractionPlan_t comp_plan;
 cudaStream_t stream;
};


CuQuantumExecutor::CuQuantumExecutor()
{
 const size_t version = cutensornetGetVersion();
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): cuTensorNet backend version " << version << std::endl;

 int num_gpus = 0;
 auto error_code = talshDeviceCount(DEV_NVIDIA_GPU,&num_gpus); assert(error_code == TALSH_SUCCESS);
 for(int i = 0; i < num_gpus; ++i){
  if(talshDeviceState(i,DEV_NVIDIA_GPU) >= DEV_ON) gpus.emplace_back(i);
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Number of available GPUs = " << gpus.size() << std::endl;

 ctn_handles.resize(gpus.size());
 for(const auto & gpu_id: gpus){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu_id));
  HANDLE_CTN_ERROR(cutensornetCreate((cutensornetHandle_t*)(&ctn_handles[gpu_id])));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Created cuTensorNet contexts for all available GPUs" << std::endl;

}


CuQuantumExecutor::~CuQuantumExecutor()
{
 bool success = sync(); assert(success);
 for(const auto & gpu_id: gpus){
  HANDLE_CUDA_ERROR(cudaSetDevice(gpu_id));
  HANDLE_CTN_ERROR(cutensornetDestroy((cutensornetHandle_t)(ctn_handles[gpu_id])));
 }
 std::cout << "#DEBUG(exatn::runtime::CuQuantumExecutor): Destroyed cuTensorNet contexts for all available GPUs" << std::endl;
 ctn_handles.clear();
 gpus.clear();
}


int CuQuantumExecutor::execute(std::shared_ptr<numerics::TensorNetwork> network,
                               TensorOpExecHandle exec_handle)
{
 int error_code = 0;
 //`Finish
 return error_code;
}


bool CuQuantumExecutor::executing(TensorOpExecHandle exec_handle)
{
 auto iter = active_networks_.find(exec_handle);
 return (iter != active_networks_.end());
}


bool CuQuantumExecutor::sync(TensorOpExecHandle exec_handle,
                             int * error_code,
                             bool wait)
{
 bool synced = true;
 *error_code = 0;
 auto iter = active_networks_.find(exec_handle);
 if(iter != active_networks_.end()){
  //`Finish
 }
 return synced;
}


bool CuQuantumExecutor::sync()
{
 bool synced = true;
 //`Finish
 return synced;
}

} //namespace runtime
} //namespace exatn

#endif //CUQUANTUM
